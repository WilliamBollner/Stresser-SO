#include <hip/hip_runtime.h>
#include <cmath>

extern "C" __global__ void stressKernel(float* data, int iterations) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < blockDim.x * gridDim.x) { // Ensure the index is valid
        float value = data[idx];
        for (int i = 0; i < iterations; ++i) {
            value = sinf(value) * cosf(value) + tanf(value);
        }
        data[idx] = value;  // Store the final result
    }
}